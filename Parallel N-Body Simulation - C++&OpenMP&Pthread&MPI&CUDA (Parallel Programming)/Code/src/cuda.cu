#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <vector>
#include <utility>
#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics_cuda.h"
#include "./headers/checkpoint.h"

int block_size = 512;

int n_body;
int n_iteration;

std::chrono::duration<double> total_time;

void generate_data(double *m, double *x, double *y, double *vx, double *vy, int n)
{
    // TODO: Generate proper initial position and mass for better visualization
    for (int i = 0; i < n; i++)
    {
        m[i] = rand() % max_mass + 1.0f;
        // Leave marginal space to make the starting position of the ball closer,
        // so as to facilitate observation of the ball movement.
        x[i] = rand() % (bound_x / 2) + (bound_x / 4);
        y[i] = rand() % (bound_y / 2) + (bound_y / 4);
        vx[i] = 0.0f;
        vy[i] = 0.0f;
    }
}

__device__ double collision(double v1, double v2, double m1, double m2)
{
    return ((m1 - m2) * v1 + 2 * m2 * v2) / (m1 + m2);
}

__global__ void update_position(double *x, double *y, double *vx, double *vy, double* new_vx, double* new_vy, int n)
{
    // TODO: update position
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
    {
        vx[i] = new_vx[i];
        vy[i] = new_vy[i];
        x[i] += vx[i] * dt_d;
        y[i] += vy[i] * dt_d;
    }
}



__global__ void update_velocity(double *m, double *x, double *y, double *vx, double *vy, double* new_vx, double* new_vy, int n)
{
    // TODO: calculate force and acceleration, update velocity
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("blockDim.x = %d, blockIdx.x = %d, threadIdx.x = %d\n", blockDim.x, blockIdx.x, threadIdx.x);

    if (i < n)
    {
        // printf("Thread %d get x[%d] = %f\n", threadIdx.x, i, x[threadIdx.x]);
        double ax = 0;
        double ay = 0;
        new_vx[i] = vx[i];
        new_vy[i] = vy[i];
        for (int j = 0; j < n; j++)
        {
            if (i == j)
                continue;
            double dx = x[j] - x[i];
            double dy = y[j] - y[i];
            double r = sqrt(dx * dx + dy * dy);
            // if (i == 0) printf("Device %d: calculate distance from %d to %d: dx = %f, dy = %f, r = %f\n", threadIdx.x, i, j, dx, dy, r);

            if (r < radius1_d)
            {
                double dax = gravity_const_d * m[j] * dx / ((r + error_d) * (r + error_d) * (r + error_d));
                double day = gravity_const_d * m[j] * dy / ((r + error_d) * (r + error_d) * (r + error_d));
                ax += dax;
                ay += day;
                // if (i == 0) printf("Device %d: calculate acc_x = %f * %f * %f / (%f + %f)^3 = %f\n", threadIdx.x, gravity_const_d, m[j], dx, r, error_d, dax);
                // if (i == 0) printf("Device %d: calculate acc_y = %f * %f * %f / (%f + %f)^3 = %f\n", threadIdx.x, gravity_const_d, m[j], dy, r, error_d, day);
            }
            else
            {
                double dax = gravity_const_d * m[j] * dx / (r * r * r);
                double day = gravity_const_d * m[j] * dy / (r * r * r);
                ax += dax;
                ay += day;
                // if (i == 0) printf("Device %d: calculate acc from %d to %d: dax = %f, day = %f\n", threadIdx.x, i, j, dax, day);
            }

            if (r < radius2_d)
            { // collision
                double v1x = new_vx[i];
                double v1y = new_vy[i];
                double v2x = vx[j];
                double v2y = vy[j];
                new_vx[i] = collision(v1x, v2x, m[i], m[j]);
                new_vy[i] = collision(v1y, v2y, m[i], m[j]);
                // if (i == 0) printf("Device %d: collision from %d to %d: v1y = %f, v11y = %f\n", threadIdx.x, i, j, v1y, new_vy[threadIdx.x]);
            }
        }
        // if (i == 0) printf("Work item %d calculated ax = %f, ay = %f\n", i, ax[threadIdx.x], ay[threadIdx.x]);

        if (x[i] <= 0 || x[i] >= bound_x_d)
        {
            new_vx[i] = -new_vx[i];
        }
        new_vx[i] += ax * dt_d;
        // Set the extreme seep to make the ball's motion visible.
        if (new_vx[i] > 20000000.0f)
            new_vx[i] = 20000000.0f;
        else if (new_vx[i] < -20000000.0f)
            new_vx[i] = -20000000.0f;

        if (y[i] <= 0 || y[i] >= bound_y_d)
        {
            new_vy[i] = -new_vy[i];
        }
        new_vy[i] += ay * dt_d;
        // Set the extreme seep to make the ball's motion visible.
        if (new_vy[i] > 20000000.0f)
            new_vy[i] = 20000000.0f;
        else if (new_vy[i] < -20000000.0f)
            new_vy[i] = -20000000.0f;
        // if (i == 0) printf("Work item %d calculated new_vx = %f, new_vy = %f\n", i, new_vx[threadIdx.x], new_vy[threadIdx.x]);
    }
}

void master()
{
    double *m = new double[n_body];
    double *x = new double[n_body];
    double *y = new double[n_body];
    double *vx = new double[n_body];
    double *vy = new double[n_body];

    generate_data(m, x, y, vx, vy, n_body);

    Logger l = Logger("cuda", n_body, bound_x, bound_y);

    double *device_m;
    double *device_x;
    double *device_y;
    double *device_vx;
    double *device_vy;

    double *device_new_vx;
    double *device_new_vy;

    hipMalloc(&device_m, n_body * sizeof(double));
    hipMalloc(&device_x, n_body * sizeof(double));
    hipMalloc(&device_y, n_body * sizeof(double));
    hipMalloc(&device_vx, n_body * sizeof(double));
    hipMalloc(&device_vy, n_body * sizeof(double));
    hipMalloc(&device_new_vx, n_body * sizeof(double));
    hipMalloc(&device_new_vy, n_body * sizeof(double));

    hipMemcpy(device_m, m, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_x, x, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_y, y, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vx, vx, n_body * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_vy, vy, n_body * sizeof(double), hipMemcpyHostToDevice);

    int n_block = n_body / block_size + 1;

    for (int i = 0; i < n_iteration; i++)
    {
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        update_velocity<<<n_block, block_size>>>(device_m, device_x, device_y, device_vx, device_vy, device_new_vx, device_new_vy, n_body);
        update_position<<<n_block, block_size>>>(device_x, device_y, device_vx, device_vy, device_new_vx, device_new_vy, n_body);

        hipMemcpy(x, device_x, n_body * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(y, device_y, n_body * sizeof(double), hipMemcpyDeviceToHost);

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> time_span = t2 - t1;
        total_time += time_span;

        l.save_frame(x, y);

        // printf("Iteration %d, elapsed time: %.3f\n", i, time_span);

#ifdef GUI
        glClear(GL_COLOR_BUFFER_BIT);
        glColor3f(1.0f, 0.0f, 0.0f);
        glPointSize(2.0f);
        glBegin(GL_POINTS);
        double xi;
        double yi;
        for (int i = 0; i < n_body; i++)
        {
            xi = x[i];
            yi = y[i];
            glVertex2f(xi, yi);
        }
        glEnd();
        glFlush();
        glutSwapBuffers();
#else

#endif
    }

    hipFree(device_m);
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_vx);
    hipFree(device_vy);
    hipFree(device_new_vx);
    hipFree(device_new_vy);

    delete m;
    delete x;
    delete y;
    delete vx;
    delete vy;
}

int main(int argc, char *argv[])
{

    n_body = atoi(argv[1]);
    n_iteration = atoi(argv[2]);

#ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(800, 800);
    glutCreateWindow("N Body Simulation CUDA Implementation");
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    gluOrtho2D(0, bound_x, 0, bound_y);
#endif

    master();

    printf("Student ID: 119010265\n"); // replace it with your student id
    printf("Name: SHI Wenlan\n");      // replace it with your name
    printf("Assignment 3: N Body Simulation CUDA Implementation\n");
    printf("n_body = %d, n_iteration = %d\n", n_body, n_iteration);
    printf("Used time: %.3f\n", total_time);

    return 0;
}
