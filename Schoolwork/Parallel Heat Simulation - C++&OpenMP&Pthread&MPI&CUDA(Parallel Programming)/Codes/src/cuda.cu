#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <algorithm>

#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"


int block_size = 512; // cuda thread block size
int size; // problem size

std::chrono::duration<double> total_time;

void initialize(float* data, bool* fire_area) {
    // TODO: intialize the temperature distribution (in parallelized way)
    int len = size * size;
    for (int i = 0; i < len; i++) {
        if (fire_area[i]) data[i] = fire_temp;
        else data[i] = wall_temp;
    }
}

void generate_fire_area(bool* fire_area) {
    // TODO: generate the fire area (in parallelized way)
    int len = size * size;
    for (int i = 0; i < len; i++) {
        fire_area[i] = 0;
    }

    float fire1_r2 = fire_size * fire_size;
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            int a = i - size / 2;
            int b = j - size / 2;
            int r2 = 0.5 * a * a + 0.8 * b * b - 0.5 * a * b;
            if (r2 < fire1_r2) fire_area[i * size + j] = 1;
        }
    }

    float fire2_r2 = (fire_size / 2) * (fire_size / 2);
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            int a = i - 1 * size / 3;
            int b = j - 1 * size / 3;
            int r2 = a * a + b * b;
            if (r2 < fire2_r2) fire_area[i * size + j] = 1;
        }
    }
}

__device__ bool shouldCal(int size, int i, bool* fire_area) {
    if (i >= (size * size - size)) return false;
    if (i < size) return false;
    if (i % size == 0) return false;
    if ((i + 1) % size == 0) return false;
    if (fire_area[i]) return false;
    return true;
}

__global__ void update(float* data, float* new_data, bool* fire_area, int size) {
    // TODO: update temperature for each point  (in parallelized way)
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (shouldCal(size, i, fire_area)) {
        float up = data[i - size];
        float down = data[i + size];
        float left = data[i - 1];
        float right = data[i + 1];
        float new_val = (up + down + left + right) / 4;
        new_data[i] = new_val;
    }
}

bool check_continue(float* data, float* new_data) {
    // TODO: determine if we should stop (because the temperature distribution will finally converge)
    int len = size * size;
    for (int i = 0; i < len; i++) {
        float gap = std::max(data[i] - new_data[i], new_data[i] - data[i]);
        //printf("index = %d, new_data = %d, gap = %f\n", i, new_data[i], gap);
        if (gap > threshold) return true;
    }
    return false;
}

#ifdef GUI
void data2pixels(float* data, GLubyte* pixels) {
    // convert rawdata (large, size^2) to pixels (small, resolution^2) for faster rendering speed
    float factor_data_pixel = (float)size / resolution;
    float factor_temp_color = (float)255 / (fire_temp - wall_temp);
    for (int x = 0; x < resolution; x++) {
        for (int y = 0; y < resolution; y++) {
            int idx = x * resolution + y;
            int idx_pixel = idx * 3;
            int x_raw = x * factor_data_pixel;
            int y_raw = y * factor_data_pixel;
            int idx_raw = y_raw * size + x_raw;
            float temp = data[idx_raw];
            int color = ((int)temp / 5 * 5 - wall_temp) * factor_temp_color;
            pixels[idx_pixel] = color;
            pixels[idx_pixel + 1] = 255 - color;
            pixels[idx_pixel + 2] = 255 - color;
        }
    }
}


void plot(GLubyte* pixels) {
    // visualize temprature distribution
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(resolution, resolution, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    glutSwapBuffers();
}
#endif


void master() {
    float* data_odd_host = new float[size * size];
    float* data_even_host = new float[size * size];
    bool* fire_area_host = new bool[size * size];

    float* data_odd;
    float* data_even;
    bool* fire_area;

    hipMalloc(&data_odd, size * size * sizeof(float));
    hipMalloc(&data_even, size * size * sizeof(float));
    hipMalloc(&fire_area, size * size * sizeof(bool));

    #ifdef GUI
    GLubyte* pixels = new GLubyte[resolution * resolution * 3];
    #endif

    generate_fire_area(fire_area_host);
    initialize(data_odd_host, fire_area_host);
    initialize(data_even_host, fire_area_host);
    //printf("After initialize, data_odd_host[1500] = %f, data_even_host[1500] = %f\n", data_odd_host[1500], data_even_host[1500]);

    hipMemcpy(data_odd, data_odd_host, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data_even, data_even_host, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(fire_area, fire_area_host, size * size * sizeof(bool), hipMemcpyHostToDevice);

    // bool cont = true;
    int count = 1;
    double total_time = 0;

    int n_block_size = size * size / block_size + 1;

    // while (cont) {
    for (int c = 0; c < 300; c++){
        std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        if (count % 2 == 1) {
            update << <n_block_size, block_size >> > (data_odd, data_even, fire_area, size);
        }
        else {
            update << <n_block_size, block_size >> > (data_even, data_odd, fire_area, size);
        }

        std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        double this_time = std::chrono::duration<double>(t2 - t1).count();
        total_time += this_time;
        // printf("Iteration %d, elapsed time: %.6f\n", count, this_time);     

        if (count % 2 == 1) {
            hipMemcpy(data_even_host, data_even, size * size * sizeof(float), hipMemcpyDeviceToHost);
            //printf("After iteration %d, data_odd_host[1500] = %f, data_even_host[1500] = %f\n", count, data_odd_host[1500], data_even_host[1500]);
            #ifdef GUI
            data2pixels(data_even_host, pixels);
            #endif
        }
        else {
            hipMemcpy(data_odd_host, data_odd, size * size * sizeof(float), hipMemcpyDeviceToHost);
            //printf("After iteration %d, data_odd_host[1500] = %f, data_even_host[1500] = %f\n", count, data_odd_host[1500], data_even_host[1500]);
            #ifdef GUI
            data2pixels(data_odd_host, pixels);
            #endif
        }

        // cont = check_continue(data_odd_host, data_even_host);
        #ifdef GUI
        plot(pixels);
        #endif

        count++;
    }

    printf("Converge after %d iterations, elapsed time: %.6f, average computation time: %.6f\n", count - 1, total_time, (double)total_time / (count - 1));

    delete data_odd_host;
    delete data_even_host;
    delete fire_area_host;
    hipFree(data_odd);
    hipFree(data_even);
    hipFree(fire_area);

    #ifdef GUI
    hipFree(pixels);
    #endif
    
}


int main(int argc, char *argv[]){
    
    size = atoi(argv[1]);

    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(resolution, resolution);
    glutCreateWindow("Heat Distribution Simulation CUDA Implementation");
    gluOrtho2D(0, resolution, 0, resolution);
    #endif

    master();

    printf("size = %d\n", size);
    printf("Student ID: 119010265\n"); // replace it with your student id
    printf("Name: SHI Wenlan\n"); // replace it with your name
    printf("Assignment 4: Heat Distribution CUDA Implementation\n");
    
    return 0;

}


