﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ void perlocateUp(node* array[], int hole) {
    int parent;
    node* tmp = array[hole];
    while (hole / 2 > 0) {
        parent = hole / 2;
        if (array[parent]->counter > tmp->counter) {
            array[hole] = array[parent];
            array[hole]->position = hole;
        }
        else {
            break;
        }
        hole = parent;
    }
    array[hole] = tmp;
    array[hole]->position = hole;
};

__device__ void perlocateDown(node* array[], int hole) {
    int child;
    node* tmp = array[hole];
    while (hole * 2 <= 1023) {
        child = hole * 2;
        if (child != 1023 && array[child + 1]->counter < array[child]->counter) {
            child++;
        }
        if (array[child]->counter < tmp->counter) {
            array[hole] = array[child];
            array[hole]->position = hole;
        }
        else {
            break;
        }
        hole = child;
    }
    array[hole] = tmp;
    array[hole]->position = hole;
};

__device__ void init_invert_page_table(VirtualMemory *vm) {
  
  for (int i = 0; i < vm->PAGE_ENTRIES; i++) {  // PAGE_ENTRIES = 1024, 4KB for each thread
    vm->invert_page_table[i] = 0x80000000 + i;   // valid bit: invalid: = 0x8000 0000, valid = 0x0, + virtual address
    vm->invert_page_table[i + vm->PAGE_ENTRIES] = 0x80000000 + i;
    vm->invert_page_table[i + 2*vm->PAGE_ENTRIES] = 0x80000000 + i;
    vm->invert_page_table[i + 3*vm->PAGE_ENTRIES] = 0x80000000 + i;

    node* new_node = new node;
    new_node->physical_address = i;
    new_node->position = i+1;
    vm->nodes[i] = new_node;
    vm->LRU[i+1] = new_node;
  }
}

__device__ void vm_init(VirtualMemory *vm, uchar *buffer, uchar *storage,
                        u32 *invert_page_table, int *pagefault_num_ptr,
                        int PAGESIZE, int INVERT_PAGE_TABLE_SIZE,
                        int PHYSICAL_MEM_SIZE, int STORAGE_SIZE,
                        int PAGE_ENTRIES) {
  // init variables
  vm->buffer = buffer;
  vm->storage = storage;
  vm->invert_page_table = invert_page_table;
  vm->pagefault_num_ptr = pagefault_num_ptr;

  // init constants
  vm->PAGESIZE = PAGESIZE;                                  // 32B
  vm->INVERT_PAGE_TABLE_SIZE = INVERT_PAGE_TABLE_SIZE;      // 16KB
  vm->PHYSICAL_MEM_SIZE = PHYSICAL_MEM_SIZE;                // 32KB
  vm->STORAGE_SIZE = STORAGE_SIZE;                          // 128KB
  vm->PAGE_ENTRIES = PAGE_ENTRIES;                          // 1024

  // before first vm_write or vm_read
  init_invert_page_table(vm);
}

__device__ uchar vm_read(VirtualMemory *vm, u32 addr) {
    int pid_adder = vm->pid * vm->PAGE_ENTRIES;
    vm->count++;
    u32 offset = addr - ((addr >> 5) << 5);
    addr = addr >> 5;
    for (int i = 0; i < 1024; i++) {    // if the data need to be read is in the buffer
        if (vm->invert_page_table[i + pid_adder] < 0x80000000 && vm->invert_page_table[i + pid_adder] == addr) {
            vm->nodes[i]->counter = vm->count;
            perlocateDown(vm->LRU, vm->nodes[i]->position);
            return vm->buffer[(i << 5) + offset];
        }
    }
    (*vm->pagefault_num_ptr)++;   // if the data need to be read is not in the buffer
    int address = vm->LRU[1]->physical_address;
    u32 vpn = vm->invert_page_table[address + pid_adder];
    for (int i = 0; i < 32; i++) {
        vm->storage[(vpn << 5) + i] = vm->buffer[(address << 5) + i];   // store page into the disk
        vm->buffer[(address << 5) + i] = vm->storage[(addr << 5) + i];  // load page from the disk
    }
    vm->invert_page_table[address + pid_adder] = addr;
    vm->LRU[1]->counter = vm->count;
    perlocateDown(vm->LRU, 1);
    return vm->buffer[(address << 5) + offset];

    /* Complate vm_read function to read single element from data buffer */
    //return 123; //TODO
}

__device__ void vm_write(VirtualMemory *vm, u32 addr, uchar value) {
    int pid_adder = vm->pid * vm->PAGE_ENTRIES;
    vm->count++;
    u32 offset = addr - ((addr >> 5) << 5);
    addr = addr >> 5;
    for (int i = 0; i < 1024; i++) {    // if the data need to be rewrite is in the buffer
        if (vm->invert_page_table[i + pid_adder] < 0x80000000 && vm->invert_page_table[i + pid_adder] == addr) {
            vm->buffer[(i << 5) + offset] = value;
            vm->nodes[i]->counter = vm->count;
            perlocateDown(vm->LRU, vm->nodes[i]->position);
            return;
        }
    }
    (*vm->pagefault_num_ptr)++;    // if it is a piece of new data or the data need to be rewrite is in the disk
    if (vm->pointer < 1024) {   // pointer point to the next physical address to be filled before physical memory filled
        vm->buffer[(vm->pointer << 5) + offset] = value;    // store the data
        vm->invert_page_table[vm->pointer + pid_adder] = addr; // set the valid bit and store the virtual address
        vm->nodes[vm->pointer]->counter = vm->count;    // update the LRU counter
        perlocateUp(vm->LRU, vm->nodes[vm->pointer]->position);
        perlocateDown(vm->LRU, vm->nodes[vm->pointer]->position);
        vm->pointer++;  // next position
    }
    else {  // swap data to disk according to LRU
        int address = vm->LRU[1]->physical_address;
        u32 vpn = vm->invert_page_table[address];
        for (int i = 0; i < 32; i++) {
            vm->storage[(vpn << 5) + i] = vm->buffer[(address << 5) + i];   // store page into the disk
            vm->buffer[(address << 5) + i] = vm->storage[(addr << 5) + i];  // load page from the disk
        }
        vm->buffer[(address << 5) + offset] = value;
        vm->invert_page_table[address + pid_adder] = addr;
        vm->LRU[1]->counter = vm->count;
        perlocateDown(vm->LRU, 1);
    }
  /* Complete vm_write function to write value into data buffer */
}

__device__ void vm_snapshot(VirtualMemory *vm, uchar *results, int offset,
                            int input_size) {
    for (int i = offset; i < input_size; i++)
        results[i] = vm_read(vm, i);
  /* Complete snapshot function togther with vm_read to load elements from data
   * to result buffer */
}

