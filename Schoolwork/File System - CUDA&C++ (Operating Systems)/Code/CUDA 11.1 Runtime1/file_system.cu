#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;


__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
	// init variables
	fs->volume = volume;

	// init constants
	fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;		// 4096, 32K/8 bits = 4 K, bit wise free space management
	fs->FCB_SIZE = FCB_SIZE;					// 32, 32 bytes per FCB
	fs->FCB_ENTRIES = FCB_ENTRIES;				// 1024
	fs->STORAGE_SIZE = VOLUME_SIZE;				// 1085440, 4096+32768+1048576
	fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;// 32
	fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;	// 20
	fs->MAX_FILE_NUM = MAX_FILE_NUM;			// 1024
	fs->MAX_FILE_SIZE = MAX_FILE_SIZE;			// 1048576
	fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;	// 36864

	for (int i = 0; i < fs->STORAGE_SIZE; i++) {	// initialize as 0
		fs->volume[i] = 0;
	} 
}


__device__ void perlocateDown_D(FCB* array[], int hole, int size) { // first order: modified time; second order: create time
	int child;
	FCB* tmp = array[hole];
	while (hole * 2 <= size) {
		child = hole * 2;
		if (child != size && array[child + 1]->modify_time > array[child]->modify_time) {
			child++;
		}
		else if (child != size && array[child + 1]->modify_time == array[child]->modify_time) {
			if (array[child + 1]->create_time < array[child]->create_time) {
				child++;
			}
		}
		if (array[child]->modify_time > tmp->modify_time) {
			array[hole] = array[child];
		}
		else if (array[child]->modify_time == tmp->modify_time) {
			if (array[child]->create_time < array[hole]->create_time) {
				array[hole] = array[child];
			}
			else {
				break;
			}
		}
		else {
			break;
		}
		hole = child;
	}
	array[hole] = tmp;
};

__device__ void perlocateDown_S(FCB* array[], int hole, int size) { // first order: size; second order: create time
	int child;
	FCB* tmp = array[hole];
	while (hole * 2 <= size) {
		child = hole * 2;
		if (child != size && array[child + 1]->size > array[child]->size) {
			child++;
		}
		else if (child != size && array[child + 1]->size == array[child]->size) {
			if (array[child + 1]->create_time < array[child]->create_time) {
				child++;
			}
		}
		if (array[child]->size > tmp->size) {
			array[hole] = array[child];
		}
		else if (array[child]->size == tmp->size) {
			if (array[child]->create_time < array[hole]->create_time) {
				array[hole] = array[child];
			}
			else {
				break;
			}
		}
		else {
			break;
		}
		hole = child;
	}
	array[hole] = tmp;
};

__device__ void perlocateDown_C(FCB* array[], int hole, int size) { 
	int child;
	FCB* tmp = array[hole];
	while (hole * 2 <= size) {
		child = hole * 2;
		if (child != size && array[child + 1]->start_address < array[child]->start_address) {
			child++;
		}
		if (array[child]->start_address < tmp->start_address) {
			array[hole] = array[child];
		}
		else {
			break;
		}
		hole = child;
	}
	array[hole] = tmp;
};

__device__ uchar bit_change(uchar ch, int i, int v) { // change the ith bit in ch to v
	int bit[9];
	bit[8] = (ch >> 7);
	bit[7] = (ch >> 6) - (bit[8] << 1);
	bit[6] = (ch >> 5) - (bit[8] << 2) - (bit[7] << 1);
	bit[5] = (ch >> 4) - (bit[8] << 3) - (bit[7] << 2) - (bit[6] << 1);
	bit[4] = (ch >> 3) - (bit[8] << 4) - (bit[7] << 3) - (bit[6] << 2) - (bit[5] << 1);
	bit[3] = (ch >> 2) - (bit[8] << 5) - (bit[7] << 4) - (bit[6] << 3) - (bit[5] << 2) - (bit[4] << 1);
	bit[2] = (ch >> 1) - (bit[8] << 6) - (bit[7] << 5) - (bit[6] << 4) - (bit[5] << 3) - (bit[4] << 2) - (bit[3] << 1);
	bit[1] = ch - ((ch >> 1) << 1);

	bit[i] = v;
	uchar result = (bit[8] << 7) + (bit[7] << 6) + (bit[6] << 5) + (bit[5] << 4) + (bit[4] << 3) + (bit[3] << 2) + (bit[2] << 1) + bit[1];
	return result;	
}

__device__ int bit_check(uchar ch, int count) { // check continuous empty blocks(0)
	int bit[9];
	bit[8] = (ch >> 7);
	bit[7] = (ch >> 6) - (bit[8] << 1);
	bit[6] = (ch >> 5) - (bit[8] << 2) - (bit[7] << 1);
	bit[5] = (ch >> 4) - (bit[8] << 3) - (bit[7] << 2) - (bit[6] << 1);
	bit[4] = (ch >> 3) - (bit[8] << 4) - (bit[7] << 3) - (bit[6] << 2) - (bit[5] << 1);
	bit[3] = (ch >> 2) - (bit[8] << 5) - (bit[7] << 4) - (bit[6] << 3) - (bit[5] << 2) - (bit[4] << 1);
	bit[2] = (ch >> 1) - (bit[8] << 6) - (bit[7] << 5) - (bit[6] << 4) - (bit[5] << 3) - (bit[4] << 2) - (bit[3] << 1);
	bit[1] = ch - ((ch >> 1) << 1);

	for (int i = 1; i < 9; i++) {
		if (bit[i] == 0) {
			count++;
		}
		else {
			count = 0;
		}
	}
	
	return count;
}

__device__ int find_space(FileSystem* fs, int size = 32) {	// find a continuous (size = 1024/32) blocks memory 
	int count = 0;
	int start = 99999;	// if find, start = 0 ~ 32768; if not find, start = 99999
	int i;
	for (i = 0; i < fs->SUPERBLOCK_SIZE; i++) {	// 0 ~ 4096-1
		if (count >= size) {	// 1024/32 = 32 storage blocks
			start = i * 8 - count;
			break;
		}
		count = bit_check(fs->volume[i], count);
		//printf("count up to now: %d\n", count);
	}
	return start;
}

__device__ void compaction(FileSystem* fs) {
	FCB* min_heap[1025];	// start address low to high, move to lower empty blocks
	int count = 0;
	//printf("Let's do a full compaction!\n");
	for (int i = 0; i < fs->FCB_ENTRIES; i++) {
		int fcb = fs->SUPERBLOCK_SIZE + i * fs->STORAGE_BLOCK_SIZE;
		if (fs->volume[fcb] != 0) {	// not empty
			FCB* new_FCB = new FCB;
			new_FCB->index = i;
			new_FCB->start_address = fs->volume[fcb + 20] + (fs->volume[fcb + 21] << 8);
			min_heap[1 + count++] = new_FCB;
		}
	}
	//printf("Process: filled min heap!\n");
	for (int i = count / 2; i > 0; i--) {	// build the min heap
		perlocateDown_C(min_heap, i, count);
	}
	//printf("Process: built min heap!\n");
	for (int j = count; j > 0; j--) {
		//printf("Process: start %d\n", j);
		int index = min_heap[1]->index;
		int address = min_heap[1]->start_address;	// block index
		int fcb = fs->SUPERBLOCK_SIZE + index * fs->STORAGE_BLOCK_SIZE;
		int size = fs->volume[fcb + 22] + (fs->volume[fcb + 23] << 8);
		int fp1 = fs->FILE_BASE_ADDRESS + address * fs->STORAGE_BLOCK_SIZE;	// old data address
		//printf("Process: identify old address of %d: %d\n", index, address);
		for (int i = 0; i < ((size + 31) / 32); i++) {	// clear free space management
			int byte = (address + i) / 8;
			int offset = (address + i) % 8 + 1;
			//printf("%d->", fs->volume[byte]);
			fs->volume[byte] = bit_change(fs->volume[byte], offset, 0);
			//printf("%d\n", fs->volume[byte]);
		}
		//printf("Process: cleared free space management, size:%d\n", ((size + 31) / 32));
		int new_address = find_space(fs, ((size + 31) / 32));
		//printf("Process: find new address: %d\n", new_address);
		if (new_address == 99999) {
			for (int i = 0; i < ((size + 31) / 32); i++) {	// modify back free space management
				int byte = (address + i) / 8;
				int offset = (address + i) % 8 + 1;
				fs->volume[byte] = bit_change(fs->volume[byte], offset, 1);
			}
			min_heap[1]->start_address = INT_MAX - 10;	// push it to the end
			perlocateDown_C(min_heap, 1, count);
			//printf("Process: skiped %d\n", j);
			continue;
		}
		int fp2 = fs->FILE_BASE_ADDRESS + new_address * fs->STORAGE_BLOCK_SIZE;	// new data address
		for (u32 i = 0; i < size; i++) {				// write in file data
			fs->volume[fp2 + i] = fs->volume[fp1 + i];
		}
		//printf("Process: write the file\n");
		for (int i = 0; i < ((size + 31) / 32); i++) {	// fill free space management
			int byte = (new_address + i) / 8;
			int offset = (new_address + i) % 8 + 1;
			fs->volume[byte] = bit_change(fs->volume[byte], offset, 1);
		}
		//printf("Process: filled free space management\n");
		fs->volume[fcb + 20] = new_address - ((new_address >> 8) << 8);	// update new start address in FCB
		fs->volume[fcb + 21] = new_address >> 8;
		min_heap[1]->start_address = INT_MAX - 10;	// push it to the end
		//printf("Process: stored new address\n");
		perlocateDown_C(min_heap, 1, count);
		//printf("Process: moved %d\n", j);
	}
	//printf("Full compaction finished!\n");
}



__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
	gtime++;
	// Check if it exist in FCB
	int fcb;	// fcb address
	int empty = 0;	// next empty fcb
	bool flag = false;	// find it or not
	u32 address;	// address to store the data = FCB[20:23] 
	for (int i = 0; i < fs->FCB_ENTRIES; i++) {	// try to find exist fcb
		fcb = fs->SUPERBLOCK_SIZE + i * fs->STORAGE_BLOCK_SIZE;
		int qointer = fcb;
		int pointer = 0;
		if (fs->volume[fcb] == 0 && empty == 0) {
			empty = fcb;
		}
		while (fs->volume[qointer] != 0 || s[pointer] != 0) {	// check the file name
			if (fs->volume[qointer] == s[pointer]) {
				//printf("i = %d: %c<->%c\n", i, fs->volume[qointer], s[pointer]);
				qointer++;
				pointer++;
				flag = true;
			}
			else {
				flag = false;
				break;
			}
		}
		if (flag == true) {
			break;
		}
	}
	if (flag == true) {	// find the file
		//printf("find the exist file at %d\n",(fcb - fs->SUPERBLOCK_SIZE)/fs->STORAGE_BLOCK_SIZE);
		address = fs->volume[fcb + 20] + (fs->volume[fcb + 21] << 8);	// 0 ~ 32768-1
		if (op == G_WRITE) {	// op == G_WRITE, need clear; op == G_READ, only need to return the address
			int size = (fs->volume[fcb + 22] + (fs->volume[fcb + 23] << 8) + 31) / 32;	// unit: storage block, 0~32
			for (int i = 0; i < size; i++) {	// clear free space management may be enough
				int byte = (address + i) / 8;
				int offset = (address + i) % 8 + 1;
				fs->volume[byte] = bit_change(fs->volume[byte], offset, 0);
			}
			address = find_space(fs);	// find another space for new write in content
			if (address >= 99999) {	// try full compaction once
				compaction(fs);
				address = find_space(fs);
			}
			if (address < 99999) {
				fs->volume[fcb + 20] = address - ((address >> 8) << 8);	// update new start address in FCB
				fs->volume[fcb + 21] = address >> 8;
				fs->volume[fcb + 22] = 0;	// clear the size in FCB
				fs->volume[fcb + 23] = 0;
				fs->volume[fcb + 28] = gtime - ((gtime >> 8) << 8);	// update modify time
				fs->volume[fcb + 29] = (gtime >> 8) - ((gtime >> 16) << 8);
				fs->volume[fcb + 30] = (gtime >> 16) - ((gtime >> 24) << 8);
				fs->volume[fcb + 31] = gtime >> 24;
				//printf("new address: %d\n", fs->volume[fcb + 20] + (fs-find_space>volume[fcb + 21] << 8));
			}
			else {
				printf("Attention: Memory is completely full!!\n");
			}
		}
	}
	else {	// New file, find a contiguous 1024 bytes of block of memory, modify the FCB
		//printf("create a new file at %d\n", (empty - fs->SUPERBLOCK_SIZE) / fs->STORAGE_BLOCK_SIZE);
		address = find_space(fs);
		if (address >= 99999) {	// try full compaction once
			compaction(fs);
			address = find_space(fs);
		}
		fcb = empty;
		if (address >= 99999) {
			printf("Attention: Memory is completely full!!\n");
		}
		else if (empty == 0) {
			printf("Attention: FCB entries is completely full!!\n");
		}
		else {
			if (op == G_READ) {
				printf("Attention: No such file to read!!\n");
			}
			else {
				int qointer = empty;	// create a new FCB
				int pointer = 0;
				while (s[pointer] != 0) {	// fill the file name in FCB
					fs->volume[qointer] = s[pointer];
					qointer++;
					pointer++;
				}
				fs->volume[qointer] = '\0';
				fs->volume[empty + 20] = address - ((address >> 8) << 8);	// update new start address in FCB
				fs->volume[empty + 21] = address >> 8;
				fs->volume[empty + 22] = 0;	// clear the size (bytes) in FCB
				fs->volume[empty + 23] = 0;
				fs->volume[empty + 24] = gtime - ((gtime >> 8) << 8);	// update create time
				fs->volume[empty + 25] = (gtime >> 8) - ((gtime >> 16) << 8);
				fs->volume[empty + 26] = (gtime >> 16) - ((gtime >> 24) << 8);
				fs->volume[empty + 27] = gtime >> 24;
				fs->volume[empty + 28] = gtime - ((gtime >> 8) << 8);	// update modify time
				fs->volume[empty + 29] = (gtime >> 8) - ((gtime >> 16) << 8);
				fs->volume[empty + 30] = (gtime >> 16) - ((gtime >> 24) << 8);
				fs->volume[empty + 31] = gtime >> 24;
			}
		}
	}
	//printf("open file at %d\n", address);
	return fcb; // return the address of fcb
}


__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	
	gtime++;
	int fcb = fp;
	int address = fs->volume[fcb + 20] + (fs->volume[fcb + 21] << 8);
	fp = fs->FILE_BASE_ADDRESS + address * fs->STORAGE_BLOCK_SIZE;
	u32 f_size = fs->volume[fcb + 22] + (fs->volume[fcb + 23] << 8);
	if (f_size < size) {
		printf("Attention: read out of bounds, only read the contents of the current file!!\n");
		size = f_size;
	}
	for (u32 i = 0; i < size; i++) {
		output[i] = fs->volume[fp+i];
	}
	//printf("read file at %d\n", address);
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	gtime++;
	int fcb = fp;
	int address = fs->volume[fcb + 20] + (fs->volume[fcb + 21] << 8);
	fp = fs->FILE_BASE_ADDRESS + address * fs->STORAGE_BLOCK_SIZE;
	for (u32 i = 0; i < size; i++) {				// write in file data
		fs->volume[fp + i] = input[i];
	}
	for (u32 i = 0; i < ((size + 31) / 32); i++) {	// update Free Space Management
		int byte = (address + i) / 8;
		int offset = (address + i) % 8 + 1;
		fs->volume[byte] = bit_change(fs->volume[byte], offset, 1);
	}
	fs->volume[fcb + 22] = size - ((size >> 8) << 8);	// update size
	fs->volume[fcb + 23] = size >> 8;
	fs->volume[fcb + 28] = gtime - ((gtime >> 8) << 8);	// update modify time
	fs->volume[fcb + 29] = (gtime >> 8) - ((gtime >> 16) << 8);
	fs->volume[fcb + 30] = (gtime >> 16) - ((gtime >> 24) << 8);
	fs->volume[fcb + 31] = gtime >> 24;
	
	//printf("write file at %d\n", address);
	return 0;
	/* Implement write operation here */
}

__device__ void fs_gsys(FileSystem *fs, int op)	// LS_D: order by modified time/ LS_S: list with size, order by size
{
	gtime++;
	FCB* min_heap[1025];
	int count = 0;
	if (op == LS_D) {
		printf("==sort by modified time==\n");
		for (int i = 0; i < fs->FCB_ENTRIES; i++) {
			int fcb = fs->SUPERBLOCK_SIZE + i * fs->STORAGE_BLOCK_SIZE;
			if (fs->volume[fcb] != 0) {
				FCB* new_FCB = new FCB;
				new_FCB->index = i;
				new_FCB->modify_time = fs->volume[fcb + 28] + (fs->volume[fcb + 29] << 8) + (fs->volume[fcb + 30] << 16) + (fs->volume[fcb + 31] << 24);
				new_FCB->create_time = fs->volume[fcb + 24] + (fs->volume[fcb + 25] << 8) + (fs->volume[fcb + 26] << 16) + (fs->volume[fcb + 27] << 24);
				min_heap[1 + count++] = new_FCB;
			}
		}
		for (int i = count / 2; i > 0; i--) {	// build the min heap
			perlocateDown_D(min_heap, i, count);
		}
		for (int i = count; i > 0; i--) {
			int qointer = fs->SUPERBLOCK_SIZE + min_heap[1]->index * fs->STORAGE_BLOCK_SIZE;
			char name[20];
			int j = 0;
			for (j = 0; j < 20; j++) {
				char ch = fs->volume[qointer + j];
				name[j] = ch;
				if (ch == 0) break;
			}
			for (; j < 20; j++) {
				name[j] = 0;
			}
			printf("%s\n", name);
			//while (fs->volume[qointer] != '\0') {	
			//	printf("%c", fs->volume[qointer++]);
			//}
			//printf("\n");
			min_heap[1]->modify_time = -1;	// push it to the end
			perlocateDown_D(min_heap, 1, count);
		}
	}
	else if (op == LS_S) {
		printf("==sort by file size==\n");
		for (int i = 0; i < fs->FCB_ENTRIES; i++) {
			int fcb = fs->SUPERBLOCK_SIZE + i * fs->STORAGE_BLOCK_SIZE;
			if (fs->volume[fcb] != 0) {
				FCB* new_FCB = new FCB;
				new_FCB->index = i;
				new_FCB->size = fs->volume[fcb + 22] + (fs->volume[fcb + 23] << 8);
				//printf("count = %d, size = %d\n", count, new_FCB->size);
				new_FCB->create_time = fs->volume[fcb + 24] + (fs->volume[fcb + 25] << 8) + (fs->volume[fcb + 26] << 16) + (fs->volume[fcb + 27] << 24);
				min_heap[1 + count++] = new_FCB;
			}

		}
		for (int i = count / 2; i > 0; i--) {	// build the min heap
			perlocateDown_S(min_heap, i, count);
		}
		for (int i = count; i > 0; i--) {
			int qointer = fs->SUPERBLOCK_SIZE + min_heap[1]->index * fs->STORAGE_BLOCK_SIZE;
			char name[20];
			int j = 0;
			for (j = 0; j < 20; j++) {
				char ch = fs->volume[qointer + j];
				name[j] = ch;
				if (ch == 0) break;
			}
			for (; j < 20; j++) {
				name[j] = 0;
			}
			printf("%s\t%d\n", name, min_heap[1]->size);
			//while (fs->volume[qointer] != 0) {	// check the file name
			//	printf("%c", fs->volume[qointer++]);
			//}
			//printf("\t%d\n", min_heap[1]->size);
			min_heap[1]->size = -1;	// push it to the end
			perlocateDown_S(min_heap, 1, count);
		}
	}
	
	/* Implement LS_D and LS_S operation here */
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)	// RM
{
	gtime++;
	int fcb;	// fcb address
	bool flag = false;	// find it or not
	for (int j = 0; j < fs->FCB_ENTRIES; j++) {	// try to find exist fcb
		fcb = fs->SUPERBLOCK_SIZE + j * fs->STORAGE_BLOCK_SIZE;
		int qointer = fcb;
		int pointer = 0;
		while (fs->volume[qointer] != 0 || s[pointer] != 0) {	// check the file name
			if (fs->volume[qointer] == s[pointer]) {
				qointer++;
				pointer++;
				flag = true;
			}
			else {
				flag = false;
				break;
			}
		}
		if (flag == true) {
			fs->volume[fcb] = 0;
			u32 address = fs->volume[fcb + 20] + (fs->volume[fcb + 21] << 8);
			u32 size = fs->volume[fcb + 22] + (fs->volume[fcb + 23] << 8);
			for (u32 i = 0; i < ((size + 31) / 32); i++) {	// update Free Space Management
				int byte = (address + i) / 8;
				int offset = (address + i) % 8 + 1;
				fs->volume[byte] = bit_change(fs->volume[byte], offset, 0);
			}
			break;
		}
	}
	if (flag == false) {
		printf("Attention: fail to find the file to remove");
	}
	/* Implement rm operation here */
}
